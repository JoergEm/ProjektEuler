//
// CUDA 11.7
// _0001multiples_of_3_or_5.cu
// 
// Copyright 2025 @GitHub JoergEm
// 
// This program is WITHOUT ANY WARRANTY; without even the implied
// warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
//

#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void _0001multiples_of_3_or_5(int* result) {

    int nummer = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (nummer < 1000) {
        if (nummer % 3 == 0 || nummer % 5 == 0) {
            atomicAdd(result, nummer);
        }
    }
}

int main() {
    int *d_result;
    int summe = 0;

    hipMalloc(&d_result, sizeof(int));
    hipMemcpy(d_result, &summe, sizeof(int), hipMemcpyHostToDevice);

    // 999 threads 1..999?
    _0001multiples_of_3_or_5<<<(999 + 255) / 256, 256>>>(d_result);

    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();

    if (errSync != hipSuccess)
        printf("Sync error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async error: %s\n", hipGetErrorString(errAsync));

    hipMemcpy(&summe, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d\n", summe);

    hipFree(d_result);

    return 0;
}